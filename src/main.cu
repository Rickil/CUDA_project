#include "hip/hip_runtime.h"
#include "image.hh"
#include "pipeline.hh"
#include "fix_cpu.cuh"
#include "fix_gpu.cuh"
#include "fix_gpu_perfect.cuh"

#include <vector>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <filesystem>
#include <numeric>

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    // -- Pipeline initialization

    std::cout << "File loading..." << std::endl;

    // - Get file paths

    using recursive_directory_iterator = std::filesystem::recursive_directory_iterator;
    std::vector<std::string> filepaths;
    for (const auto& dir_entry : recursive_directory_iterator("../images"))
        filepaths.emplace_back(dir_entry.path());

    // - Init pipeline object

    Pipeline pipeline(filepaths);

    // -- Main loop containing image retring from pipeline and fixing

    const int nb_images = pipeline.images.size();
    std::vector<Image> images(nb_images);

    // - One CPU thread is launched for each image

    std::cout << "Done, starting compute" << std::endl;

    std::cout << "START\n";
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);

    hipEventRecord(gpu_start);

    // CPU timer
    clock_t cpu_start, cpu_end;
    cpu_start = clock();

    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        // TODO: Make it GPU compatible (aka faster)
        // You will need to copy images one by one on the GPU
        // You can store the images the way you want on the GPU
        // But you should treat the pipeline as a pipeline:
        // You *must not* copy all the images and only then do the computations
        // You must get the image from the pipeline as they arrive and launch computations right away
        // There are still ways to speed up this process, of course (wait for the last class)

        // i = 20;
        // printf("image: %d\n", i);
        images[i] = pipeline.get_image(i);
        fix_image_gpu(images[i]);
        /*images[i] = pipeline.get_image(i);
        fix_image_cpu(images[i]);*/
        // break;
    }

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);

    float gpu_milliseconds = 0;
    hipEventElapsedTime(&gpu_milliseconds, gpu_start, gpu_stop);

    cpu_end = clock();
    float cpu_milliseconds = 1000.0f * (cpu_end - cpu_start) / CLOCKS_PER_SEC;

    std::cout << "Done with compute, starting stats" << std::endl;
    std::cout << "GPU Time: " << gpu_milliseconds << " milliseconds" << std::endl;
    std::cout << "CPU Time: " << cpu_milliseconds << " milliseconds" << std::endl;
    std::cout << "GPU vs CPU Time: " << gpu_milliseconds << " ms (GPU) vs " << cpu_milliseconds << " ms (CPU)" << std::endl;


    /*double seconds = milliseconds / 1000.0;
    double fps = images.size() / seconds;

    std::cout << "FPS: " << fps << std::endl;*/

    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);


    // -- All images are now fixed : compute stats (total then sort)

    // - First compute the total of each image

    // TODO : make it GPU compatible (aka faster)
    // You can use multiple CPU threads for your GPU version using openmp or not
    // Up to you :)
    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        auto& image = images[i];
        const int image_size = image.width * image.height;
        image.to_sort.total = std::reduce(image.buffer, image.buffer + image_size, 0);
    }

    // - All totals are known, sort images accordingly (OPTIONAL)
    // Moving the actual images is too expensive, sort image indices instead
    // Copying to an id array and sort it instead

    // TODO OPTIONAL : for you GPU version you can store it the way you want
    // But just like the CPU version, moving the actual images while sorting will be too slow
    using ToSort = Image::ToSort;
    std::vector<ToSort> to_sort(nb_images);
    std::generate(to_sort.begin(), to_sort.end(), [n = 0, images] () mutable
    {
        return images[n++].to_sort;
    });

    // TODO OPTIONAL : make it GPU compatible (aka faster)
    std::sort(to_sort.begin(), to_sort.end(), [](ToSort a, ToSort b) {
        return a.total < b.total;
    });

    // TODO : Test here that you have the same results
    // You can compare visually and should compare image vectors values and "total" values
    // If you did the sorting, check that the ids are in the same order
    for (int i = 0; i < nb_images; ++i)
    {
        std::cout << "Image #" << images[i].to_sort.id << " total : " << images[i].to_sort.total << std::endl;
        std::ostringstream oss;
        oss << "Image#" << images[i].to_sort.id << ".pgm";
        std::string str = oss.str();
        images[i].write(str);
    }

    std::cout << "Done, the internet is safe now :)" << std::endl;

    // Cleaning
    // TODO : Don't forget to update this if you change allocation style
    for (int i = 0; i < nb_images; ++i)
        free(images[i].buffer);

    return 0;
}
